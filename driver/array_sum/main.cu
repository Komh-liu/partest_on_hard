#include "array_sum.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <chrono>
#include <cstdlib> // 用于 exit()

std::vector<long long> load_array_from_file(const std::string& filename) {
    Array arr;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(1);
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        long long num;
        while (iss >> num) {
            arr.push_back(num);
        }
    }

    file.close();
    return arr;
}

long long load_result_from_file(const std::string& filename) {
    long long result;
    std::ifstream file(filename);
    std::string line;

    if (!file.is_open()) {
        std::cerr << "无法打开文件: " << filename << std::endl;
        exit(1);
    }

    while (std::getline(file, line)) {
        std::istringstream iss(line);
        iss >> result;
    }

    file.close();
    return result;
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "用法: " << argv[0] << " <数组文件路径> <结果文件路径>" << std::endl;
        return 1;
    }

    // 从命令行参数获取文件路径
    std::string data_file_path = argv[1];
    std::string result_file_path = argv[2];

    // 从文件加载数组
    std::vector<long long> arr = load_array_from_file(data_file_path);
    long long result = load_result_from_file(result_file_path);

    // 记录开始时间戳并写入标准输出
    auto time_start = std::chrono::high_resolution_clock::now();
    auto start_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
        time_start.time_since_epoch()
    ).count();
    std::cout << "[METRICS] ARRAY_TIME_START=" << start_ms << std::endl << std::flush;

    // 调用CUDA加速的数组求和函数
    long long sum = array_sum(arr);
    std::cout << "数组的和是: " << sum << std::endl;

    // 记录结束时间戳并写入标准输出
    auto time_end = std::chrono::high_resolution_clock::now();
    auto end_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
        time_end.time_since_epoch()
    ).count();
    std::cout << "[METRICS] ARRAY_TIME_END=" << end_ms << std::endl << std::flush;
    
    // 输出耗时和验证结果
    std::cout << "Time: " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count()
            << "ms\n";

    if (result == sum)
        std::cout << "验证成功" << std::endl;
    else
        std::cout << "验证失败" << std::endl;

    return 0;
}
