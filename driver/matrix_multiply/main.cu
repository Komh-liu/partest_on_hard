#include "matrix_multiply.h"
#include <iostream>
#include <chrono>
#include <fstream>
#include <vector>
#include <string>
#include <ctime>
#include <filesystem>

// CUDA版本使用一维数组表示矩阵，这里需要加载为一维形式
Matrix load_matrix(const std::string& filename, int& N, int& M) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(1);
    }

    int rows, cols;
    file >> rows >> cols; // 读取矩阵的行数和列数
    N = rows;
    M = cols;

    // 初始化为全零矩阵（一维表示）
    Matrix matrix(rows * cols, 0);

    // 读取三元组形式的非零元素
    int i, j, value;
    while (file >> i >> j >> value) {
        if (i >= 0 && i < rows && j >= 0 && j < cols) {
            matrix[i * cols + j] = value;
        } else {
            std::cerr << "Invalid matrix coordinates: (" << i << ", " << j << ")" << std::endl;
        }
    }

    return matrix;
}

// 将一维矩阵保存为三元组形式
void save_matrix(const Matrix& matrix, int rows, int cols, const std::string& filename) {
    std::ofstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(1);
    }

    // 保存非零元素为三元组形式
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            int index = i * cols + j;
            if (matrix[index] != 0) {
                file << i << " " << j << " " << matrix[index] << std::endl;
            }
        }
    }
}

// 比较两个文本文件是否相同
bool compare_text_files(const std::string& file1, const std::string& file2) {
    // 打开两个文件
    std::ifstream f1(file1);
    std::ifstream f2(file2);

    // 检查文件是否成功打开
    if (!f1.is_open() || !f2.is_open()) {
        std::cerr << "无法打开文件！" << std::endl;
        return false;
    }

    // 逐行读取并比较
    std::string line1, line2;
    bool same = true;

    while (true) {
        std::getline(f1, line1);
        std::getline(f2, line2);

        // 检查是否同时到达文件末尾
        if (f1.eof() && f2.eof()) {
            break;
        }

        // 如果一个文件到达末尾而另一个没有，行数不同
        if (f1.eof() != f2.eof()) {
            std::cerr << "文件行数不同！" << std::endl;
            same = false;
            break;
        }

        // 比较当前行
        if (line1 != line2) {
            std::cerr << "文件内容不同！" << std::endl;
            same = false;
            break;
        }
    }
    return same;
}

// 生成包含时间戳的文件名
std::string generate_filename_with_timestamp(const std::string& base_filename) {
    // 获取当前时间
    std::time_t now = std::time(nullptr);
    std::tm* now_tm = std::localtime(&now);

    // 格式化时间戳
    char timestamp_str[20];
    std::strftime(timestamp_str, sizeof(timestamp_str), "%Y%m%d_%H%M%S", now_tm);

    // 提取文件名（不包含路径）
    std::filesystem::path path(base_filename);
    std::string filename = path.filename().string();

    // 生成新文件名
    std::string new_filename = filename + "_" + timestamp_str + ".txt";

    // 返回新文件的完整路径
    return (path.parent_path() / new_filename).string();
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <input_matrix_file> <output_result_file>" << std::endl;
        return 1;
    }

    std::string input_file = argv[1];
    std::string output_file = argv[2];

    // 加载矩阵（一维表示）
    int N, M;  // 矩阵的行数和列数
    Matrix A = load_matrix(input_file, N, M);
    Matrix result(M * M, 0);  // 初始化结果矩阵为全零（一维表示）

    // 记录开始时间戳并写入标准输出
    auto time_start = std::chrono::high_resolution_clock::now();
    auto start_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
        time_start.time_since_epoch()
    ).count();
    std::cout << "[METRICS] MATRIX_TIME_START=" << start_ms << std::endl << std::flush;

    // 执行矩阵乘法
    matrix_multiply(A, N, M, result);  // 使用CUDA特定的函数签名

    // 记录结束时间戳并写入标准输出
    auto time_end = std::chrono::high_resolution_clock::now();
    auto end_ms = std::chrono::duration_cast<std::chrono::milliseconds>(
        time_end.time_since_epoch()
    ).count();
    std::cout << "[METRICS] MATRIX_TIME_END=" << end_ms << std::endl << std::flush;

    // 输出耗时
    std::cout << "Time: " 
            << std::chrono::duration_cast<std::chrono::milliseconds>(time_end - time_start).count()
            << "ms\n";

    // 生成包含时间戳的文件名
    std::string combined_file = generate_filename_with_timestamp(output_file);
    save_matrix(result, M, M, combined_file);  // 保存结果矩阵

    // 比较结果
    bool c_result = compare_text_files(combined_file, output_file);
    if(c_result)
       std::cout<<"验证成功"<<std::endl;
    else
        std::cout<<"验证失败"<<std::endl;
    
    return 0;
}
